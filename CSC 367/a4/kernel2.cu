#include "hip/hip_runtime.h"
/* ------------
 * This code is provided solely for the personal and private use of
 * students taking the CSC367H5 course at the University of Toronto.
 * Copying for purposes other than this use is expressly prohibited.
 * All forms of distribution of this code, whether as given or with
 * any changes, are expressly prohibited.
 *
 * Authors: Bogdan Simion, Felipe de Azevedo Piovezan
 *
 * All of the files in this directory and all subdirectories are:
 * Copyright (c) 2022 Bogdan Simion
 * -------------
 */

#include "kernels.h"
#include <stdio.h>

void run_kernel2(const int8_t *filter, int32_t dimension, const int32_t *input,
                 int32_t *output, int32_t width, int32_t height, float * transfer_in_time, float *computation_time ,float *transfer_out_time) {  
  int total_num_threads, num_pixels;
  num_pixels = total_num_threads = width * height;

  //Every block created will have 1024 threads. Minimum number of block is 1.
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  int num_of_thread_per_block = deviceProp.maxThreadsDim[0];
  int num_block = (total_num_threads/num_of_thread_per_block<=0) ? 1 : (total_num_threads+num_of_thread_per_block-1)/num_of_thread_per_block;
  //These are all variables and array passed to a kernel, they will be allocated on gpu memory 
  int *max_array_device;
  int *min_array_device;
  int8_t *filter_device;
  int32_t *input_device;
  int32_t *output_device;

  //each block stores 2 values local max and local min on max_array and min_array.
  int *max_array_cpu;
  int *min_array_cpu;

  //cudamalloc to malloc memory on device memory. 
  hipError_t cudaStatus;
  size_t SIZE = num_pixels * sizeof(int32_t);
  size_t SIZE_filter = sizeof(int8_t) * dimension * dimension;
  size_t SIZE_max_min_array = sizeof(int)*num_block;

  int global_max_cpu;
  int global_min_cpu;
  
  //Allocate space on gpu memory for device variable and array
  cudaStatus = hipMalloc((void**)&input_device, SIZE);
  if (cudaStatus != hipSuccess) 
  {
      fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
  }

  cudaStatus = hipMalloc((void**)&output_device, SIZE);
  if (cudaStatus != hipSuccess) 
  {
      fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
  }

  max_array_cpu = (int *) malloc(SIZE_max_min_array);
  min_array_cpu = (int *) malloc(SIZE_max_min_array);

  cudaStatus = hipMalloc((void**)&max_array_device, SIZE_max_min_array); 
  if (cudaStatus != hipSuccess) 
  {
      fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
  }
  
  cudaStatus = hipMalloc((void**)&min_array_device, SIZE_max_min_array); 
  if (cudaStatus != hipSuccess) 
  {
      fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
  }

  cudaStatus = hipMalloc((void**)&filter_device, SIZE_filter); 
  if (cudaStatus != hipSuccess) 
  {
      fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
  }

  //transfer in event
  //input_device and filter_device array will to contain value from input and filter.
  //So kernel can use these value stored on gpu

  //calculate transfer in time
  hipEvent_t transfer_in_start, transfer_in_stop;
  hipEventCreate(&transfer_in_start);
  hipEventCreate(&transfer_in_stop);
  hipEventRecord(transfer_in_start);

  cudaStatus = hipMemcpy(input_device, input, SIZE, hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) 
  {
      fprintf(stderr, "hipMemcpy fail: %s\n", hipGetErrorString(cudaStatus));
  }

  cudaStatus = hipMemcpy(filter_device, filter, SIZE_filter, hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) 
  {
      fprintf(stderr, "hipMemcpy fail: %s\n", hipGetErrorString(cudaStatus));
  }

  hipEventRecord(transfer_in_stop);
  hipEventSynchronize(transfer_in_stop);
  hipEventElapsedTime(transfer_in_time, transfer_in_start, transfer_in_stop);

  //computation event 
  //Process is very similar to kernel 1
  hipEvent_t computation_start_1, computation_stop_1;
  float computation_time_1;
  hipEventCreate(&computation_start_1);
  hipEventCreate(&computation_stop_1);
  hipEventRecord(computation_start_1);

  kernel2<<<num_block, num_of_thread_per_block>>>(filter_device, dimension, input_device, output_device, width, height, max_array_device, min_array_device);
  hipDeviceSynchronize();

  hipEventRecord(computation_stop_1);
  hipEventSynchronize(computation_stop_1);
  hipEventElapsedTime(&computation_time_1, computation_start_1, computation_stop_1);

  //transfer out event
  //Copied computed result stored in output array to output on cpu
  //Similar to computation event, this event is divided into two parts as well
  hipEvent_t transfer_out_start_1, transfer_out_stop_1;
  float transfer_out_time_1;
  hipEventCreate(&transfer_out_start_1);
  hipEventCreate(&transfer_out_stop_1);
  hipEventRecord(transfer_out_start_1);

  cudaStatus = hipMemcpy(max_array_cpu, max_array_device, sizeof(int)*num_block, hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy fail: %s\n", hipGetErrorString(cudaStatus));
  }
  cudaStatus = hipMemcpy(min_array_cpu, min_array_device, sizeof(int)*num_block, hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy fail: %s\n", hipGetErrorString(cudaStatus));
  }

  hipEventRecord(transfer_out_stop_1);
  hipEventSynchronize(transfer_out_stop_1);
  hipEventElapsedTime(&transfer_out_time_1, transfer_out_start_1, transfer_out_stop_1);

  //Second part of computation. Finding global max and min and normalization.
  hipEvent_t computation_start_2, computation_stop_2;
  float computation_time_2;
  hipEventCreate(&computation_start_2);
  hipEventCreate(&computation_stop_2);
  hipEventRecord(computation_start_2);

  //compute global max and global min
  global_max_cpu = max_array_cpu[0];
  global_min_cpu = min_array_cpu[0];
  for (int i = 0; i < num_block; i++) 
  {
    if (max_array_cpu[i]>global_max_cpu) 
    {
      global_max_cpu = max_array_cpu[i];
    }
    if (min_array_cpu[i]<global_min_cpu) 
    {
      global_min_cpu = min_array_cpu[i];
    }
  }

  //normalize using global max cpu and global min cpu
  normalize2<<<num_block, num_of_thread_per_block>>>(output_device, width, height, global_max_cpu, global_min_cpu);
  hipDeviceSynchronize();
  hipEventRecord(computation_stop_2);
  hipEventSynchronize(computation_stop_2);
  hipEventElapsedTime(&computation_time_2, computation_start_2, computation_stop_2);

  //Transfer out final result to output
  hipEvent_t transfer_out_start_2, transfer_out_stop_2;
  float transfer_out_time_2;
  hipEventCreate(&transfer_out_start_2);
  hipEventCreate(&transfer_out_stop_2);
  hipEventRecord(transfer_out_start_2);

  cudaStatus = hipMemcpy(output, output_device, SIZE, hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy fail: %s\n", hipGetErrorString(cudaStatus));
  }

  hipEventRecord(transfer_out_stop_2);
  hipEventSynchronize(transfer_out_stop_2);
  hipEventElapsedTime(&transfer_out_time_2, transfer_out_start_2, transfer_out_stop_2);

  //Add up computation time and transfer out time
  *computation_time = computation_time_1 + computation_time_2;
  *transfer_out_time = transfer_out_time_1 + transfer_out_time_2;

  //Free gpu and cpu memory
  hipFree(input_device);
  hipFree(output_device);
  hipFree(max_array_device);
  hipFree(min_array_device);
  hipFree(filter_device);
  free(max_array_cpu);
  free(min_array_cpu);
}

/* Processes a single pixel and returns the value of processed pixel */
__device__ int32_t apply2d(const int8_t *f, int32_t dimension, const int32_t *input, int row, int column,
                int32_t width, int32_t height) 
{
  int val = 0;
  int filter_ind = 0;
  int offset = (dimension-1)/2;
  int start_row_ind = row-offset;
  int start_col_ind = column-offset;
  int end_row_ind = row-offset+dimension;
  int end_col_ind = column-offset+dimension;
  for (int i = start_row_ind; i < end_row_ind; i++) 
  {
    for (int j = start_col_ind; j < end_col_ind; j++) 
    {
      if(0 <= i && i < height && 0 <= j && j < width) {
        val += f[filter_ind] * input[i * width + j];
      }
      filter_ind++;
    }
  }
  return val;
}

/* Utilize multiple threads to do computation */
__global__ void kernel2(const int8_t *filter, int32_t dimension,
                        const int32_t *input, int32_t *output, int32_t width,
                        int32_t height, int *max_array, int *min_array) {
  
  int global_tid = blockDim.x * blockIdx.x + threadIdx.x;
  int local_tid = threadIdx.x;
	__shared__ int local_max_array[1024];
	__shared__ int local_min_array[1024];

  //need to know number of active thread
  __shared__ int activeThreadCounter;

  if (threadIdx.x == 0) {
      activeThreadCounter = 0;
  }
  __syncthreads();

  //Calculate total number of threads which should actually do work 
  if(global_tid < width*height) {
      atomicAdd(&activeThreadCounter, 1);
  }
  __syncthreads();  
  
  //Notice this kernel is row major 
  int row_ind, col_ind;
  col_ind = global_tid%width;
  row_ind = global_tid/width;

  int32_t processed_pixel = apply2d(filter, dimension, input, row_ind, col_ind, width, height);
  __syncthreads();

  if (global_tid < width * height) 
  {
    output[row_ind * width + col_ind] = processed_pixel;
  }
  local_max_array[local_tid] = processed_pixel;
  local_min_array[local_tid] = processed_pixel;
  __syncthreads();

  //Start local reduction on 1024 values or less in a block
  //and store max and min value within a block in max_min_array 
  //based on block id
	for (unsigned int s = 1; s < activeThreadCounter; s *= 2) 
  {
		if(local_tid % (2*s) == 0 && local_max_array[local_tid] < local_max_array[local_tid + s] && local_tid + s < activeThreadCounter)  
    { 
      local_max_array[local_tid] = local_max_array[local_tid + s];
    }
    else if(local_tid % (2*s) == 0 && local_min_array[local_tid] > local_min_array[local_tid + s] && local_tid + s < activeThreadCounter)  
    {
      local_min_array[local_tid] = local_min_array[local_tid + s];
		}
		__syncthreads();
	}

  //local_max_array and local_min_array have local max 
  //and local min at index 0 respectively
  max_array[blockIdx.x] = local_max_array[0];
  min_array[blockIdx.x] = local_min_array[0];
  __syncthreads();
}

/* Normalizes a pixel given the smallest and largest integer values
 * in the image */
__device__ void normalize_pixel(int32_t *output, int32_t pixel_idx, int32_t smallest,
                     int32_t largest) 
{
  if (smallest == largest) {
    return;
  }
  output[pixel_idx] =
      ((output[pixel_idx] - smallest) * 255) / (largest - smallest);
}

/* Similar to kernel1, but access pattern is row major */
__global__ void normalize2(int32_t *output, int32_t width, int32_t height,
                           int32_t global_max, int32_t global_min) 
{
  int global_tid = blockDim.x * blockIdx.x + threadIdx.x;
  int row_ind, col_ind;
  col_ind = global_tid%width;
  row_ind = global_tid/width;
  int pixel_ind = row_ind * width + col_ind;
  normalize_pixel(output, pixel_ind, global_min, global_max);
}
