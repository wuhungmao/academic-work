#include "hip/hip_runtime.h"
 /* ------------
 * This code is provided solely for the personal and private use of
 * students taking the CSC367H5 course at the University of Toronto.
 * Copying for purposes other than this use is expressly prohibited.
 * All forms of distribution of this code, whether as given or with
 * any changes, are expressly prohibited.
 *
 * Authors: Bogdan Simion, Felipe de Azevedo Piovezan
 *
 * All of the files in this directory and all subdirectories are:
 * Copyright (c) 2022 Bogdan Simion
 * -------------
 */

#include "kernels.h"

__global__ void dot_kernel2(float *g_idata1, float *g_idata2, float *g_odata) {

	extern __shared__ float sdata[];
	
	unsigned int tid = threadIdx.x;
	
	// Global thread id
	unsigned int i = blockIdx.x*blockDim.x + threadIdx.x;

	sdata[tid] = g_idata1[i] * g_idata2[i];
	__syncthreads();

	// do reduction in shared memory
	for (unsigned int s = 1; s < blockDim.x; s *= 2) {
		int idx = 2 * s * tid; // change this to reduce divergence
		if (idx < blockDim.x) { // In a warp, all threads participate (or don't)
			sdata[idx] += sdata[idx + s];
		} 
		__syncthreads();
	}

	// write result for this block back to global memory
	if (tid == 0) { g_odata[blockIdx.x] = sdata[0]; }
}
