#include "hip/hip_runtime.h"
/* ------------
 * This code is provided solely for the personal and private use of
 * students taking the CSC367H5 course at the University of Toronto.
 * Copying for purposes other than this use is expressly prohibited.
 * All forms of distribution of this code, whether as given or with
 * any changes, are expressly prohibited.
 *
 * Authors: Bogdan Simion, Felipe de Azevedo Piovezan
 *
 * All of the files in this directory and all subdirectories are:
 * Copyright (c) 2022 Bogdan Simion
 * -------------
 */

#include "kernels.h"

__global__ void dot_kernel4(float *g_idata1, float *g_idata2, float *g_odata) {
    
	extern __shared__ float sdata[];
	
	unsigned int tid = threadIdx.x;
	
	// Global thread id
	unsigned int i = blockIdx.x * (blockDim.x*2) + threadIdx.x;

	sdata[tid] = g_idata1[i] * g_idata2[i] + g_idata1[i + blockDim.x] * g_idata2[i + blockDim.x];
	__syncthreads();

    // do reduction in shared memory
    for (unsigned int s = blockDim.x/2; s > 0; s >>= 1) { 
        if (tid < s) {  
            sdata[tid] += sdata[tid + s];
        }
        __syncthreads();
    }
	// write result for this block back to global memory
	if (tid == 0) { g_odata[blockIdx.x] = sdata[0]; }
}
