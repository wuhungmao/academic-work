#include "hip/hip_runtime.h"
/* ------------
 * This code is provided solely for the personal and private use of
 * students taking the CSC367H5 course at the University of Toronto.
 * Copying for purposes other than this use is expressly prohibited.
 * All forms of distribution of this code, whether as given or with
 * any changes, are expressly prohibited.
 *
 * Authors: Bogdan Simion, Felipe de Azevedo Piovezan
 *
 * All of the files in this directory and all subdirectories are:
 * Copyright (c) 2022 Bogdan Simion
 * -------------
 */

#include "kernels.h"
#include <stdio.h>

void run_kernel4(const int8_t *filter, int32_t dimension, const int32_t *input,
                 int32_t *output, int32_t width, int32_t height, float * transfer_in_time, float *computation_time ,float *transfer_out_time) {
  int num_pixels = width * height;

  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  int num_of_thread_per_block = deviceProp.maxThreadsDim[0];
  int num_block = 32768;

  //stride over total number of threads
  int stride_size = num_block * num_of_thread_per_block;

  //each block stores 2 values local max and local min on max_array and min_array
  int *max_array_device;
  int *min_array_device;
  int8_t *filter_device;
  int32_t *input_device;
  int32_t *output_device;

  int *max_array_cpu;
  int *min_array_cpu;

  //cudamalloc to malloc memory on device memory. 
  hipError_t cudaStatus;
  size_t SIZE = num_pixels * sizeof(int32_t);
  size_t SIZE_filter = sizeof(int8_t) * dimension * dimension;
  size_t SIZE_max_min_array = sizeof(int)*num_block;

  int global_max_cpu;
  int global_min_cpu;

  cudaStatus = hipMalloc((void**)&input_device, SIZE);
  if (cudaStatus != hipSuccess) 
  {
      fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
  }

  cudaStatus = hipMalloc((void**)&output_device, SIZE);
  if (cudaStatus != hipSuccess) 
  {
      fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
  }

  max_array_cpu = (int *) malloc(SIZE_max_min_array);
  min_array_cpu = (int *) malloc(SIZE_max_min_array);
  
  cudaStatus = hipMalloc((void**)&max_array_device, SIZE_max_min_array); 
  if (cudaStatus != hipSuccess) 
  {
      fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
  }
  
  cudaStatus = hipMalloc((void**)&min_array_device, SIZE_max_min_array); 
  if (cudaStatus != hipSuccess) 
  {
      fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
  }

  cudaStatus = hipMalloc((void**)&filter_device, SIZE_filter); 
  if (cudaStatus != hipSuccess) 
  {
      fprintf(stderr, "hipMalloc failed: %s\n", hipGetErrorString(cudaStatus));
  }

  //transfer in event
  hipEvent_t transfer_in_start, transfer_in_stop;
  hipEventCreate(&transfer_in_start);
  hipEventCreate(&transfer_in_stop);
  hipEventRecord(transfer_in_start);

  cudaStatus = hipMemcpy(input_device, input, SIZE, hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) 
  {
      fprintf(stderr, "hipMemcpy fail: %s\n", hipGetErrorString(cudaStatus));
  }

  cudaStatus = hipMemcpy(filter_device, filter, SIZE_filter, hipMemcpyHostToDevice);
  if (cudaStatus != hipSuccess) 
  {
      fprintf(stderr, "hipMemcpy fail: %s\n", hipGetErrorString(cudaStatus));
  }

  hipEventRecord(transfer_in_stop);
  hipEventSynchronize(transfer_in_stop);
  hipEventElapsedTime(transfer_in_time, transfer_in_start, transfer_in_stop);

  //computation event 
  hipEvent_t computation_start_1, computation_stop_1;
  float computation_time_1;
  hipEventCreate(&computation_start_1);
  hipEventCreate(&computation_stop_1);
  hipEventRecord(computation_start_1);

  kernel4<<<num_block, num_of_thread_per_block>>>(filter_device, dimension, input_device, output_device, width, height, max_array_device, min_array_device, stride_size);
  hipDeviceSynchronize();

  hipEventRecord(computation_stop_1);
  hipEventSynchronize(computation_stop_1);
  hipEventElapsedTime(&computation_time_1, computation_start_1, computation_stop_1);

  //transfer out event
  //Copied computed result stored in output array to output on cpu
  hipEvent_t transfer_out_start_1, transfer_out_stop_1;
  float transfer_out_time_1;
  hipEventCreate(&transfer_out_start_1);
  hipEventCreate(&transfer_out_stop_1);
  hipEventRecord(transfer_out_start_1);

  cudaStatus = hipMemcpy(max_array_cpu, max_array_device, sizeof(int)*num_block, hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy fail: %s\n", hipGetErrorString(cudaStatus));
  }
  cudaStatus = hipMemcpy(min_array_cpu, min_array_device, sizeof(int)*num_block, hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy fail: %s\n", hipGetErrorString(cudaStatus));
  }

  hipEventRecord(transfer_out_stop_1);
  hipEventSynchronize(transfer_out_stop_1);
  hipEventElapsedTime(&transfer_out_time_1, transfer_out_start_1, transfer_out_stop_1);

  hipEvent_t computation_start_2, computation_stop_2;
  float computation_time_2;
  hipEventCreate(&computation_start_2);
  hipEventCreate(&computation_stop_2);
  hipEventRecord(computation_start_2);
  
  //compute global max and global min
  global_max_cpu = max_array_cpu[0];
  global_min_cpu = min_array_cpu[0];
  for (int i = 0; i < num_block; i++) {
    if (max_array_cpu[i]>global_max_cpu) 
    {
      global_max_cpu = max_array_cpu[i];
    }
    if (min_array_cpu[i]<global_min_cpu) 
    {
      global_min_cpu = min_array_cpu[i];
    }
  }

  //normalize using global max cpu and global min cpu
  normalize4<<<num_block, num_of_thread_per_block>>>(output_device, width, height, global_max_cpu, global_min_cpu, stride_size);
  hipEventRecord(computation_stop_2);
  hipEventSynchronize(computation_stop_2);
  hipEventElapsedTime(&computation_time_2, computation_start_2, computation_stop_2);

  hipEvent_t transfer_out_start_2, transfer_out_stop_2;
  float transfer_out_time_2;
  hipEventCreate(&transfer_out_start_2);
  hipEventCreate(&transfer_out_stop_2);
  hipEventRecord(transfer_out_start_2);
  
  cudaStatus = hipMemcpy(output, output_device, SIZE, hipMemcpyDeviceToHost);
  if (cudaStatus != hipSuccess) {
    fprintf(stderr, "hipMemcpy fail: %s\n", hipGetErrorString(cudaStatus));
  }

  hipEventRecord(transfer_out_stop_2);
  hipEventSynchronize(transfer_out_stop_2);
  hipEventElapsedTime(&transfer_out_time_2, transfer_out_start_2, transfer_out_stop_2);

  *computation_time = computation_time_1 + computation_time_2;
  *transfer_out_time = transfer_out_time_1 + transfer_out_time_2;

  //Free allocated space
  hipFree(input_device);
  hipFree(output_device);
  hipFree(max_array_device);
  hipFree(min_array_device);
  hipFree(filter_device);
  free(max_array_cpu);
  free(min_array_cpu);
}

/* Processes a single pixel and returns the value of processed pixel */
__device__ int32_t apply2d(const int8_t *f, int32_t dimension, const int32_t *input, int row, int column,
                int32_t width, int32_t height) 
{
  int val = 0;
  int filter_ind = 0;
  int offset = (dimension-1)/2;
  int start_row_ind = row-offset;
  int start_col_ind = column-offset;
  int end_row_ind = row-offset+dimension;
  int end_col_ind = column-offset+dimension;
  for (int i = start_row_ind; i < end_row_ind; i++) 
  {
    for (int j = start_col_ind; j < end_col_ind; j++) 
    {
      if(0 <= i && i < height && 0 <= j && j < width) {
        val += f[filter_ind] * input[i * width + j];
      }
      filter_ind++;
    }
  }
  return val;
}

/* Process pixels with stride */
__global__ void kernel4(const int8_t *filter, int32_t dimension,
                        const int32_t *input, int32_t *output, int32_t width,
                        int32_t height, int *max_array, int *min_array, int stride_size) 
{
  int global_tid = blockDim.x * blockIdx.x + threadIdx.x;
  int local_tid = threadIdx.x;
  int start_pixel_ind = global_tid;

  //Similar to kernel 3
	__shared__ int local_max_array[1024];
	__shared__ int local_min_array[1024];

  __shared__ int activeThreadCounter;

  if (threadIdx.x == 0) {
      activeThreadCounter = 0;
  }
  __syncthreads();

  //Find max and min value along with processing pixels
  int row_ind, col_ind, loc_max, loc_min;
  int local_array_ind = 0;
  loc_max = 0;
  loc_min = 0;
  //process through every pixel from start index and store result in local_array
  for(int i = start_pixel_ind; i < width * height; i+=stride_size) 
  {
    col_ind = i%width;
    row_ind = i/width;
    output[i] = apply2d(filter, dimension, input, row_ind, col_ind, width, height);
    __syncthreads();
    if (output[i]> loc_max)
    {
      loc_max = output[i];
    }
    if (output[i] < loc_min)
    {
      loc_min = output[i];
    }
    local_array_ind++;
    __syncthreads();
  }


  if (start_pixel_ind < width * height) 
  {
    atomicAdd(&activeThreadCounter, 1);
    __syncthreads();  

    //store loc_max and loc_min into local_max_array and local_min_array
    local_max_array[local_tid] = loc_max;
    local_min_array[local_tid] = loc_min;
    __syncthreads();

    //Each thread inside a block participate in this reduction
    //process to find out maximum and minimum value of all pixels
    //computed by every thread of the block 
    for (unsigned int s = 1; s < activeThreadCounter; s *= 2) 
    {
      if(local_tid % (2*s) == 0 && local_max_array[local_tid] < local_max_array[local_tid + s] && local_tid + s < activeThreadCounter)  
      { 
        local_max_array[local_tid] = local_max_array[local_tid + s];
      }
      else if(local_tid % (2*s) == 0 && local_min_array[local_tid] > local_min_array[local_tid + s] && local_tid + s < activeThreadCounter)  
      {
        local_min_array[local_tid] = local_min_array[local_tid + s];
      }
      __syncthreads();
    }
    
    max_array[blockIdx.x] = local_max_array[0];
    min_array[blockIdx.x] = local_min_array[0];
    __syncthreads();

  }

}

/* Normalizes a pixel given the smallest and largest integer values
 * in the image */
__device__ void normalize_pixel(int32_t *output, int32_t pixel_idx, int32_t smallest,
                     int32_t largest) 
{
  if (smallest == largest) {
    return;
  }
  output[pixel_idx] =
      ((output[pixel_idx] - smallest) * 255) / (largest - smallest);
}

/* Every thread normalize the chunk of pixel assigned to it. */
__global__ void normalize4(int32_t *output, int32_t width, int32_t height,
                           int32_t global_max, int32_t global_min, int stride_size) 
{
  int row_ind, col_ind, pixel_ind;
  int global_tid = blockDim.x * blockIdx.x + threadIdx.x;
  int start_pixel_ind = global_tid;

  for(int i = start_pixel_ind; i < width * height; i+=stride_size) 
  {
    col_ind = i%width;
    row_ind = i/width;
    pixel_ind = row_ind * width + col_ind;
    normalize_pixel(output, pixel_ind, global_min, global_max);
  }
}
